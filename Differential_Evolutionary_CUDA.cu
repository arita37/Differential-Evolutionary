#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace thrust;

#include <stdio.h>
#include <time.h>
#include <fstream>

#include "Utilities.cuh"

#define pi 3.14159265358979f

#define BLOCK_SIZE_POP	32
#define BLOCK_SIZE_RAND	64
#define BLOCK_SIZE_UNKN 8
#define BLOCK_SIZE		256

//#define DEBUG

// --- REFERENCES
//     [1] R. Storn and K. Price, “Differential evolution – a simple and efficient heuristic for global optimization over continuous spaces,” 
//     Journal of Global Optimization, vol. 11, no. 4, pp. 341–359, 1997

//     [2] Lucas de P. Veronese and Renato A. Krohling, “Differential Evolution Algorithm on the GPU with C-CUDA,” 
//     Proc. of the IEEE Congress on Evolutionary Computation, Barcelona, Spain, Jul. 18-23, 2010, pp. 1-7.

// Conventions: the index j addresses the population member while the index i addresses the member component
//              the homologous host and device variables have the same name with a "h_" or "d_" prefix, respectively
//				the __host__ and __device__ functions pointer parameters have the same name for comparison purposes. it is up to the caller to use 
//				host or device pointers, as appropriate

/****************************************/
/* EVALUATION OF THE OBJECTIVE FUNCTION */
/****************************************/
__global__ void curand_setup_kernel(hiprandState * __restrict state, const unsigned long int seed)
{
    int tid =  blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(seed, tid, 0, &state[tid]);
}

/********************************/
/* INITIALIZE POPULATION ON GPU */
/********************************/
__global__ void initialize_population_GPU(float * __restrict pop, const float * __restrict minima, const float * __restrict maxima, 
	                                      hiprandState * __restrict state, const int D, const int Np) {
	
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	
	if ((i < D) && (j < Np)) pop[j*D+i] = (maxima[i] - minima[i]) * hiprand_uniform(&state[j*D+i]) + minima[i];
}

/****************************************/
/* EVALUATION OF THE OBJECTIVE FUNCTION */
/****************************************/
__host__ __device__ float functional(const float * __restrict x, const int D) {

	float sum = 0.f;

	// --- De Jong function
	//for (int i=0; i<D; i++) sum = sum + x[i] * x[i];
	// --- Rosenbrock's saddle
	sum = 0.f;
	for (int i=1; i<D; i++) sum = sum + 100.f * (x[i] - x[i-1] * x[i-1]) * (x[i] - x[i-1] * x[i-1]) + (x[i-1] - 1.f) * (x[i-1] - 1.f);

	return sum;
}

/********************************/
/* POPULATION EVALUATION ON GPU */
/********************************/
__global__ void evaluation_GPU(const int Np, const int D, const float * __restrict pop, float * __restrict fobj) {

	int j = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (j < Np)  fobj[j] = functional(&pop[j*D], D);
}

/**********************************************************/
/* GENERATE MUTATION INDICES AND CROSS-OVER VALUES ON GPU */
/**********************************************************/
__global__ void generate_mutation_indices_and_crossover_values_GPU(float * __restrict Rand, int * __restrict mutation, const int Np, const int D,
	                                                               hiprandState * __restrict state) {
	
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	int a, b, c;
	
	if (j < Np) {

		do a=Np*(hiprand_uniform(&state[j*D]));	while(a==j);
		do b=Np*(hiprand_uniform(&state[j*D]));	while(b==j||b==a);
		do c=Np*(hiprand_uniform(&state[j*D]));	while(c==j||c==a||c==b);
		mutation[j*3]=a;
		mutation[j*3+1]=b;
		mutation[j*3+2]=c;

		Rand[j]=hiprand_uniform(&state[j*D]);
	}
}

/**********************************/
/* GENERATION OF A NEW POPULATION */
/**********************************/
__global__ void generation_new_population_GPU(const float * __restrict pop, const int NP, const int D, float * __restrict npop, const float F, 
	                                          const float CR, const float * __restrict rand, const int * __restrict mutation, 
											  const float * __restrict minimum, const float * __restrict maximum) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	
	if ((i < D) && (j < NP)) {

		// --- Mutation indices
		int a=mutation[j*3];
		int b=mutation[j*3+1];
		int c=mutation[j*3+2];

		// --- Mutation and crossover
		// --- One of the best strategies. Try F = 0.7 and CR = 0.5 as a first guess.
		if(rand[j]<CR)	npop[j*D+i] = pop[a*D+i]+F*(pop[b*D+i]-pop[c*D+i]);
		else			npop[j*D+i] = pop[j*D+i];

		// --- Other possible approaches to mutation and crossover
		// --- Not bad, but found several optimization problems where misconvergence occurs.
		//npop[j*D+i] = pop[best_old_gen_ind*D+i] + F*(pop[b*D+i]-pop_old[c*D+i]);
		// --- One of the best strategies. Try F = 0.85 and CR = 1. In case of misconvergence, try to increase NP. If this doesn't help,
		//     play around with all the control variables.
		//npop[j*D+i] = pop[j*D+i] + F*(pop[best_old_gen_ind*D+i] - pop[j*D+i]) + F*(pop[a*D+i]-pop[b*D+i]);
		// --- Powerful strategy worth trying.
		//npop[j*D+i] = pop[best_old_gen_ind*D+i] + (pop[a*D+i]+pop[b*D+i]-pop[c*D+i]-pop[d*D+i])*F;
		// --- Robust optimizer for many functions.
		//npop[j*D+i] = pop[e*D+i] + (pop[a*D+i]+pop[b*D+i]-pop[c*D+i]-pop[d*D+i])*F;

		// --- Saturation due to constraints on the unknown parameters
		if		(npop[j*D+i]>maximum[i])	npop[j*D+i]=maximum[i];
		else if	(npop[j*D+i]<minimum[i])	npop[j*D+i]=minimum[i];

	}
	
}

/*******************************/
/* POPULATION SELECTION ON GPU */
/*******************************/
// Assumption: all the optimization variables are associated to the same thread block
__global__ void selection_and_evaluation_GPU(const int Np, const int D, float * __restrict pop, const float * __restrict npop, float * __restrict fobj) {

	int i = threadIdx.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	
	if ((i < D) && (j < Np)) {

		float nfobj = functional(&npop[j*D], D);

		float temp = fobj[j];

		if (nfobj < temp) { 
			pop[j*D+i]	= npop[j*D+i];
			fobj[j]		= nfobj;
		}
	}
}

/***********************/
/* FIND MINIMUM ON GPU */
/***********************/
void find_minimum_GPU(const int N, float *t, float * __restrict minval, int * __restrict index) {
    
	// --- Wrap raw pointer with a device_ptr 
    device_ptr<float> dev_ptr = device_pointer_cast(t);

	// --- Use device_ptr in thrust min_element
	device_ptr<float> min_ptr = thrust::min_element(dev_ptr, dev_ptr + N);

	index[0] = &min_ptr[0] - &dev_ptr[0];
	
	minval[0] = min_ptr[0];;

}

/********/
/* MAIN */
/********/
int main()
{
	// --- Number of individuals in the population (Np >=4 for mutation purposes)
	int			Np		= 80;  
	// --- Dimensionality of each individual (number of unknowns)
	int			D		= 5;
	// --- Mutation factor (0 < F <= 2). Typically chosen in [0.5, 1], see Ref. [1]
	float		F		= 0.7f;
	// --- Maximum number of generations
	int			Gmax	= 2000;
	// --- Crossover constant (0 < CR <= 1)
	float		CR		= 0.4f;

	// --- Mutually different random integer indices selected from {1, 2, … ,Np}
	int *d_mutation,			// --- Device side mutation vector
		*d_best_index,			// --- Device side current optimal member index
		*h_best_index_dev;		// --- Host side current optimal member index of device side

	float *d_pop,				// --- Device side population
	*d_npop,					// --- Device side new population (trial vectors)
	*d_Rand,					// --- Device side crossover rand vector (uniformly distributed in (0,1))
	*d_fobj,					// --- Device side objective function value
	*d_maxima,					// --- Device side maximum constraints vector
	*d_minima,					// --- Device side minimum constraints vector
	*h_pop_dev_res,				// --- Host side population result of GPU computations
	*h_best_dev,				// --- Host side population best value history of device side
	*h_maxima,					// --- Host side maximum constraints vector
	*h_minima;					// --- Host side minimum constraints vector

	hiprandState *devState;		// --- Device side random generator state vector

	// --- Device side memory allocations
	gpuErrchk(hipMalloc((void**)&d_pop,D*Np*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_npop,D*Np*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_Rand,Np*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_fobj,Np*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_mutation,3*Np*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_maxima,D*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&d_minima,D*sizeof(float)));
	gpuErrchk(hipMalloc((void**)&devState, D*Np*sizeof(hiprandState)));

	// --- Host side memory allocations
	h_pop_dev_res		= (float*)malloc(D*Np*sizeof(float));
	h_best_dev			= (float*)malloc(Gmax*sizeof(float));
	h_best_index_dev	= (int*)malloc(Gmax*sizeof(int));
	h_maxima			= (float*)malloc(D*sizeof(float));
	h_minima			= (float*)malloc(D*sizeof(float));

	// --- Define grid sizes
	int Num_Blocks_Pop		= iDivUp(Np,BLOCK_SIZE_POP);
	int Num_Blocks_Rand2	= iDivUp(Np,BLOCK_SIZE_RAND);
	dim3 Grid(iDivUp(D,BLOCK_SIZE_UNKN),iDivUp(Np,BLOCK_SIZE_POP));
	dim3 Block(BLOCK_SIZE_UNKN,BLOCK_SIZE_POP);

	// --- Set maxima and minima
	for (int i=0; i<D; i++) {
		h_maxima[i] =  2.;
		h_minima[i] = -2.;
	}
	gpuErrchk(hipMemcpy(d_maxima, h_maxima, D*sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_minima, h_minima, D*sizeof(float), hipMemcpyHostToDevice));
	
	// --- Initialize cuRAND states
	curand_setup_kernel<<<iDivUp(D*Np, BLOCK_SIZE), BLOCK_SIZE>>>(devState, time(NULL));
	
	// --- Initialize popultion
	initialize_population_GPU<<<Grid, Block>>>(d_pop, d_minima, d_maxima, devState, D, Np);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	// --- Evaluate population
	evaluation_GPU<<<iDivUp(Np, BLOCK_SIZE), BLOCK_SIZE>>>(Np, D, d_pop, d_fobj);
#ifdef DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	int a, b, c;
	for(int i=0;i<Gmax;i++) {
		
		// --- Generate mutation indices and cross-over uniformly distributed random vector
		generate_mutation_indices_and_crossover_values_GPU<<<Num_Blocks_Rand2,BLOCK_SIZE_RAND>>>(d_Rand, d_mutation, Np, D, devState);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// --- Generate new population
		generation_new_population_GPU<<<Grid,Block>>>(d_pop, Np, D, d_npop, F, CR, d_Rand, d_mutation, d_minima, d_maxima);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		// --- Select new population and evaluate it
		selection_and_evaluation_GPU<<<Grid,Block>>>(Np, D, d_pop, d_npop, d_fobj);
#ifdef DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

		find_minimum_GPU(Np, d_fobj, &h_best_dev[i], &h_best_index_dev[i]);
		
		printf("Iteration: %i; best member value: %f: best member index: %i\n", i, h_best_dev[i], h_best_index_dev[i]);

	}

	gpuErrchk(hipMemcpy(h_pop_dev_res, d_pop, Np*sizeof(float), hipMemcpyDeviceToHost));
	for (int i=0; i<D; i++) printf("Variable nr. %i = %f\n", i, h_pop_dev_res[h_best_index_dev[Gmax-1]*D+i]);

	return 0;
}

